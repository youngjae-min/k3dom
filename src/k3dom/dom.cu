#include "common/common.h"
#include "common/cuda_utils.h"
#include "k3dom/dom.h"
#include "common/dom_types.h"

#include "common/kernel/init_new_particles.h"
#include "k3dom/kernel/mass_update.h"

#include <thrust/sort.h>
#include <thrust/transform.h>

#include <hip/hip_runtime.h>

#include <cmath>
#include <vector>

namespace dom
{

DOM::DOM(const Params& params)
    : DOM_c(params)
{
}

DOM::~DOM()
{
}

void DOM::updateGrid(float t, std::vector<float>& measurements_x,
                    std::vector<float>& measurements_y, std::vector<float>& measurements_z)
{
    if (updated_time > 0) { // skip the first time w/ updated_time = -1.0f
        float dt = t - updated_time;
        particlePrediction(dt);
        particleAssignment();
        gridCellOccupancyUpdate(dt, measurements_x, measurements_y, measurements_z);
        updatePersistentParticles();
        initializeNewParticles();
        statisticalMoments();
        resampling();
        
        // particle_array_test.copy(particle_array, hipMemcpyDeviceToHost);
        particle_array = particle_array_next;
    }
    else {initializeParticles(measurements_x, measurements_y, measurements_z);}

    CHECK_ERROR(hipDeviceSynchronize());

    updated_time = t;
}

void DOM::initializeParticles(std::vector<float>& measurements_x,
                              std::vector<float>& measurements_y, std::vector<float>& measurements_z)
{
    /// reflect measurements first without any assigned particles
    gridCellOccupancyUpdate(0.0f, measurements_x, measurements_y, measurements_z);
    
    CHECK_ERROR(hipGetLastError());

    thrust::device_vector<float> particle_orders_accum(grid_cell_count);
    accumulate(born_masses_array, particle_orders_accum);
    float* particle_orders_array_accum = thrust::raw_pointer_cast(particle_orders_accum.data());

    float new_weight = 1.0f / particle_count;

    normalize_particle_orders(particle_orders_array_accum, grid_cell_count, particle_count);

    initParticlesKernel1<<<grid_map_grid, block_dim>>>(grid_cell_array, particle_array,
                                                       particle_orders_array_accum, grid_cell_count);

    CHECK_ERROR(hipGetLastError());

    initParticlesKernel2<<<particles_grid, block_dim>>>(
        particle_array, grid_cell_array, rng_states, params.init_max_velocity, params.particle_min_vel, grid_size, new_weight, particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOM::gridCellOccupancyUpdate(float dt, std::vector<float>& measurements_x,
                                  std::vector<float>& measurements_y, std::vector<float>& measurements_z)
{
    meas_len = measurements_x.size();
    assert(meas_len == measurements_y.size() && meas_len == measurements_z.size());

    CHECK_ERROR(hipMalloc(&meas_x, meas_len * sizeof(float)));
    CHECK_ERROR(hipMalloc(&meas_y, meas_len * sizeof(float)));
    CHECK_ERROR(hipMalloc(&meas_z, meas_len * sizeof(float)));
    CHECK_ERROR(hipMemcpy(meas_x, measurements_x.data(), meas_len * sizeof(float), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(meas_y, measurements_y.data(), meas_len * sizeof(float), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(meas_z, measurements_z.data(), meas_len * sizeof(float), hipMemcpyHostToDevice));
    
    gridCellPredictionUpdateKernel<<<grid_map_grid, block_dim>>>(grid_cell_array, particle_array,
                                                                 born_masses_array,
                                                                 params.birth_prob, grid_cell_count,
                                                                 meas_x, meas_y, meas_z, meas_len, dt,
                                                                 grid_size, grid_size_z, params.resolution,
                                                                 params.sigma, params.ls, params.gamma, params.mass_scale,
                                                                 sensor_pos_x - center_pos_x, sensor_pos_y - center_pos_y,
                                                                 sensor_pos_z - center_pos_z);
    
    CHECK_ERROR(hipGetLastError());

    CHECK_ERROR(hipFree(meas_x));
    CHECK_ERROR(hipFree(meas_y));
    CHECK_ERROR(hipFree(meas_z));
}

} /* namespace dom */
